#include "hip/hip_runtime.h"
//
// Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//

#include "DemandGeometryViewer.h"

#include <OptiXToolkit/ShaderUtil/vec_math.h>

#include <optix.h>

#include <vector_functions.h>

using namespace otk;  // for vec_math operators

namespace demandGeometryViewer {

extern "C" static __constant__ Params g_params;

template <typename T>
__forceinline__ __device__ T* getSbtData()
{
    return reinterpret_cast<T*>( optixGetSbtDataPointer() );
}

template <typename T>
__forceinline__ __device__ uint_t& attr( T& val )
{
    return reinterpret_cast<uint_t&>( val );
}

#define float3Attr( vec_ ) attr( ( vec_ ).x ), attr( ( vec_ ).y ), attr( ( vec_ ).z )

__forceinline__ __device__ uchar4 makeColor( const float3& c )
{
    return make_uchar4( static_cast<unsigned char>( clamp( c.x, 0.0f, 1.0f ) * 255.0f ),
                        static_cast<unsigned char>( clamp( c.y, 0.0f, 1.0f ) * 255.0f ),
                        static_cast<unsigned char>( clamp( c.z, 0.0f, 1.0f ) * 255.0f ), 255u );
}

extern "C" __global__ void __raygen__pinHoleCamera()
{
    const uint3  idx    = optixGetLaunchIndex();
    const auto*  camera = getSbtData<CameraData>();
    const uint_t pixel  = g_params.width * idx.y + idx.x;

    float2 d         = make_float2( idx.x, idx.y ) / make_float2( g_params.width, g_params.height ) * 2.f - 1.f;
    float3 rayOrigin = camera->eye;
    float3 rayDir    = normalize( d.x * camera->U + d.y * camera->V + camera->W );
    float3 result{};

    float         tMin         = 0.0f;
    float         tMax         = 1e16f;
    float         rayTime      = 0.0f;
    OptixRayFlags flags        = OPTIX_RAY_FLAG_NONE;
    uint_t        sbtOffset    = RAYTYPE_RADIANCE;
    uint_t        sbtStride    = RAYTYPE_COUNT;
    uint_t        missSbtIndex = RAYTYPE_RADIANCE;
    optixTrace( g_params.traversable, rayOrigin, rayDir, tMin, tMax, rayTime, OptixVisibilityMask( 255 ), flags,
                sbtOffset, sbtStride, missSbtIndex, float3Attr( result ) );

    g_params.image[pixel] = makeColor( result );
}

static __forceinline__ __device__ void setRayPayload( float3 p )
{
    optixSetPayload_0( __float_as_uint( p.x ) );
    optixSetPayload_1( __float_as_uint( p.y ) );
    optixSetPayload_2( __float_as_uint( p.z ) );
}

extern "C" __global__ void __miss__backgroundColor()
{
    const auto* data = getSbtData<MissData>();
    setRayPayload( make_float3( data->background.x, data->background.y, data->background.z ) );
}

}  // namespace demandGeometryViewer

namespace demandGeometry {
namespace app {

__device__ Context& getContext()
{
    return demandGeometryViewer::g_params.demandGeomContext;
}

__device__ const demandLoading::DeviceContext& getDeviceContext()
{
    return demandGeometryViewer::g_params.demandContext;
}

__device__ void reportClosestHitNormal( float3 ffNormal )
{
    // Color the proxy faces by a solid color per face.
    const float3* colors = demandGeometryViewer::g_params.proxyFaceColors;
    uint_t        index{};
    if( ffNormal.x > 0.5f )
        index = 0;
    else if( ffNormal.x < -0.5f )
        index = 1;
    else if( ffNormal.y > 0.5f )
        index = 2;
    else if( ffNormal.y < -0.5f )
        index = 3;
    else if( ffNormal.z > 0.5f )
        index = 4;
    else if( ffNormal.z < -0.5f )
        index = 5;

    demandGeometryViewer::setRayPayload( colors[index] );
}

}  // namespace app
}  // namespace demandGeometry

#include <OptiXToolkit/DemandGeometry/ProxyInstancesImpl.h>
