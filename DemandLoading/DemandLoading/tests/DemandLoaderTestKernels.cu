#include "hip/hip_runtime.h"
//
// Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//

#include "DemandLoaderTestKernels.h"

#include <OptiXToolkit/Error/cudaErrorCheck.h>

#include <OptiXToolkit/DemandLoading/Texture2D.h>

using namespace demandLoading;

__global__ static void pageRequester( DeviceContext context, unsigned int pageId, bool* isResident, unsigned long long* pageTableEntry )
{
    *pageTableEntry = pagingMapOrRequest( context, pageId, isResident );
}

__host__ void launchPageRequester( hipStream_t stream, const DeviceContext& context, unsigned int pageId, bool* devIsResident, unsigned long long* pageTableEntry )
{
    pageRequester<<<1, 1, 0U, stream>>>( context, pageId, devIsResident, pageTableEntry );
    OTK_ERROR_CHECK( hipStreamSynchronize( stream ) );
    OTK_ERROR_CHECK( hipGetLastError() );
}


__global__ static void pageBatchRequester( DeviceContext context, unsigned int pageBegin, unsigned int pageEnd, PageTableEntry* pageTableEntries )
{
    unsigned int numPages = pageEnd - pageBegin;
    unsigned int index    = blockIdx.x * blockDim.x + threadIdx.x;
    if( index >= numPages )
        return;
    unsigned int pageId = pageBegin + index;

    bool           isResident;
    PageTableEntry entry = pagingMapOrRequest( context, pageId, &isResident );
    if( isResident )
    {
        pageTableEntries[index] = entry;
    }
}

__host__ void launchPageBatchRequester( hipStream_t stream, const DeviceContext& context, unsigned int pageBegin, unsigned int pageEnd, PageTableEntry* pageTableEntries )
{
    unsigned int threadsPerBlock = 32;
    unsigned int numPages        = pageEnd - pageBegin;
    unsigned int numBlocks       = ( numPages + threadsPerBlock - 1 ) / threadsPerBlock;

    // The DeviceContext is passed by value to the kernel, so it is copied to device memory when the kernel is launched.
    pageBatchRequester<<<numBlocks, threadsPerBlock, 0U, stream>>>( context, pageBegin, pageEnd, pageTableEntries );
}
