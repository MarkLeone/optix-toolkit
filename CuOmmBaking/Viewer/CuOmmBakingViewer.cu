#include "hip/hip_runtime.h"
//
// Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//

#include "Procedural.h"
#include "LaunchParams.h"
#include "CuOmmBakingAppDeviceUtil.h"

using namespace ommBakingApp;

//------------------------------------------------------------------------------
// Params - globally visible struct
//------------------------------------------------------------------------------

extern "C" {
__constant__ Params params;
}

//------------------------------------------------------------------------------
// Ray Payload - per ray data for OptiX programs
//------------------------------------------------------------------------------

struct RayPayload
{
    float3 color; // return color
    float  alpha; // return alpha
};

//------------------------------------------------------------------------------
// OptiX programs
//------------------------------------------------------------------------------
 
extern "C" __global__ void __raygen__rg()
{
    uint2 px = getPixelIndex( params.num_devices, params.device_idx );
    if( !pixelInBounds( px, params.image_width, params.image_height ) )
        return;

    // Ray for an orthographic view facing in the -z direction
    float3 origin;
    origin.x = params.eye.x + params.view_dims.x * (0.5f + px.x - 0.5f * params.image_width) / (float)params.image_width;
    origin.y = params.eye.y + params.view_dims.y * (0.5f + px.y - 0.5f * params.image_height) / ( float )params.image_height;
    origin.z = params.eye.z;
    float3 direction = make_float3( 0.0f, 0.0f, -1.0f );

    // Ray payload
    RayPayload payload;
    payload.color = make_float3( 0.f, 0.f, 0.f );
    payload.alpha = 1.f;
    
    // Trace the ray
    float tmin = 0.0f;
    float tmax = 1e16f;
    float time = 0.0f;
    traceRay( params.traversable_handle, RAY_TYPE_RADIANCE, origin, direction, tmin, tmax, time, &payload );

    float4 color = make_float4( payload.color.x, payload.color.y, payload.color.z, payload.alpha );

    // Put the final color in the result buffer
    params.result_buffer[px.y * params.image_width + px.x] = make_color( color );
}

extern "C" __global__ void __miss__ms()
{
    MissData* missData = reinterpret_cast<MissData*>( optixGetSbtDataPointer() );

    RayPayload* payload = getRayPayload();
    float3 color = payload->color;
    float alpha  = payload->alpha;

    // Mix color with background
    payload->color = color + alpha * missData->background_color;
}

__device__ float4 eval()
{
    // Evaluate the texture at the current intersection

    HitGroupData* hitData = reinterpret_cast< HitGroupData* >( optixGetSbtDataPointer() );

    float2 uv = optixGetTriangleBarycentrics();

    uint3 idx3 = hitData->indices[optixGetPrimitiveIndex()];
    float2 uv0 = hitData->texCoords[idx3.x];
    float2 uv1 = hitData->texCoords[idx3.y];
    float2 uv2 = hitData->texCoords[idx3.z];

    uv = uv0 + uv.x * ( uv1 - uv0 ) + uv.y * ( uv2 - uv0 );

    float4 color;
    if( hitData->texture_id )
    {
        color = tex2D<float4>( hitData->texture_id, uv.x, uv.y );
    }
    else
    {
        float alpha = eval_procedural<float>( { uv.x, uv.y } );
        color = { alpha, alpha, alpha, alpha };
    }

    return color;
}

extern "C" __global__ void __anyhit__ah()
{
    HitGroupData* hitData = reinterpret_cast< HitGroupData* >( optixGetSbtDataPointer() );

    float4 color = eval();

    RayPayload* payload = getRayPayload();
    if( params.visualize_omm )
    {
        payload->color = make_float3( 1.0f, 0.f, 0.4f );
        payload->alpha = 0.f;
    }
    else
    {
        payload->color += payload->alpha * color.w * make_float3( color.x, color.y, color.z );
        payload->alpha *= ( 1 - color.w );
    }

    optixIgnoreIntersection();
}

extern "C" __global__ void __closesthit__ch()
{
    float4 color = eval();

    RayPayload* payload = getRayPayload();
    payload->color = make_float3( color.x, color.y, color.z );
    payload->alpha = 0.f;
}
